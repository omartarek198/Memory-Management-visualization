#include "hip/hip_runtime.h"
﻿#include<stdio.h>
#include<process.h>
#include "hip/hip_runtime.h"
#include ""
#include<conio.h>
__global__ void Add(int* a, int* p, int npart, int* indT, int* index, int* temp, int w)
{

    temp[0] = 9999;
    int j;
    j = threadIdx.x;
    if (a[j] >= p[w])
    {
        if (a[j] - p[w] < temp[0])
        {
            temp[0] = a[j] - p[w];
            index[0] = j;
        }
    }
    if (j == npart-1)
    {
        a[index[0]] -= p[w];
        p[w] = -1;
        printf("%d", a[index[0]]);
    }
}
/*__global__ void Add2(int* a, int* p, int npart, int* indT, int* index, int* temp, int w)
{

        int j;
        j = threadIdx.x;
        if (a[index[0]] == a[w])
        {
            index[0] = w;
            break;
        }
        a[index[0]] -= p[w];
        p[w] = -1;
}*/
/*
    
    
*/

int main()
{
    bool flag = false;
    int* a, * p, i, j, npro, npart, * temp, * index, * indT, * nn;
    printf("Enter no of Partitions.\n");
    

    scanf("%d", &npart);
    hipMallocManaged(&a, npart * sizeof(int));
    for (i = 0;i < npart;i++)
    {
        printf("Enter the %dst Partition size:", i+1);
        scanf("%d", &a[i]);
    }
    printf("Enter no of Process.\n");
    scanf("%d", &npro);
    hipMallocManaged(&p, npro * sizeof(int));
    for (i = 0;i < npro;i++)
    {
        printf("Enter the size of %dst Processes:", i+1);
        scanf("%d", &p[i]);
    }
    hipMallocManaged(&temp, 1 * sizeof(int));
    hipMallocManaged(&indT, 1 * sizeof(int));
    hipMallocManaged(&index, 1 * sizeof(int));
    for (int w = 0;w < npro;w++)
    {
        Add << <1, npart >> > (a, p, npart, indT, index, temp, w);
        hipDeviceSynchronize();

        printf("Process allocated to");
        printf("%d\n", index[0]);
        /*Add2 << <1, npart >> > (a, p, npart, indT, index, temp, w);
        hipDeviceSynchronize();*/

    }
    for (i = 0;i < npro;i++)
    {
        if (p[i] != -1)
        {
            printf("Process %d must wait\n", i + 1);
        }
    }
    return 0;
    ///////////////////////////npart/npro/i/j/a[]/p[]/indT/index/temp/////////
}
